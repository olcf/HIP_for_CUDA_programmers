#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------------------------------
This program will fill 2 NxN matrices with random numbers, compute a matrix multiply on the CPU 
and then on the GPU, compare the values for correctness, and print _SUCCESS_ (if successful).

Written by Tom Papatheodore
------------------------------------------------------------------------------------------------*/

#include <stdio.h>
#include <essl.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include <complex.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
        printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
        exit(0);                                                                            \
    }                                                                                     \
}while(0)

#define N 512

typedef std::complex<double> complexd ;

int main(int argc, char *argv[])
{

    // Set device to GPU 0
    cudaErrorCheck( hipSetDevice(0) );

    /* Allocate memory for A, B, C on CPU ----------------------------------------------*/
    complexd *A = (complexd*)malloc(N*N*sizeof(complexd));
    complexd *B = (complexd*)malloc(N*N*sizeof(complexd));
    complexd *C = (complexd*)malloc(N*N*sizeof(complexd));

    /* Set Values for A, B, C on CPU ---------------------------------------------------*/

    // Max size of random double
    double max_value = 10.0;

    // Set A, B, C
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            A[i*N + j] = complexd((double)rand()/(double)(RAND_MAX/max_value) , (double)rand()/(double)(RAND_MAX/max_value));
            B[i*N + j] = complexd((double)rand()/(double)(RAND_MAX/max_value) , (double)rand()/(double)(RAND_MAX/max_value));
            C[i*N + j] = complexd(0.0 , 0.0);
        }
    }

    /* Allocate memory for d_A, d_B, d_C on GPU ----------------------------------------*/
    hipDoubleComplex *d_A, *d_B, *d_C;
    cudaErrorCheck( hipMalloc(&d_A, N*N*sizeof(hipDoubleComplex)) );
    cudaErrorCheck( hipMalloc(&d_B, N*N*sizeof(hipDoubleComplex)) );
    cudaErrorCheck( hipMalloc(&d_C, N*N*sizeof(hipDoubleComplex)) );

    /* Copy host arrays (A,B,C) to device arrays (d_A,d_B,d_C) -------------------------*/
    cudaErrorCheck(hipMemcpy(d_A, A, N*N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    cudaErrorCheck(hipMemcpy(d_B, B, N*N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    cudaErrorCheck(hipMemcpy(d_C, C, N*N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );	

    /* Perform Matrix Multiply on CPU --------------------------------------------------*/

    const complexd alpha = complexd(1.0, 1.0);
    const complexd beta = complexd(0.0, 0.0);

    const hipDoubleComplex cualpha = make_hipDoubleComplex(1.0, 1.0);
    const hipDoubleComplex cubeta = make_hipDoubleComplex(0.0, 0.0);

    zgemm("n", "n", N, N, N, alpha, A, N, B, N, beta, C, N);

    /* Perform Matrix Multiply on GPU --------------------------------------------------*/

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasStatus_t status = cublasZgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &cualpha, d_A, N, d_B, N, &cubeta, d_C, N);
    if (status != HIPBLAS_STATUS_SUCCESS){
        printf("hipblasZgemm failed with code %d\n", status);
        return EXIT_FAILURE;
    }

	/* Copy values of d_C back from GPU and compare with values calculated on CPU ------*/

    // Copy values of d_C (computed on GPU) into host array C_fromGPU	
    complexd *C_fromGPU = (complexd*)malloc(N*N*sizeof(complexd));	
    cudaErrorCheck( hipMemcpy(C_fromGPU, d_C, N*N*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    // Check if CPU and GPU give same results
    complexd tolerance = complexd(1.0e-13, 1.0e-13);
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            if(abs((C[i*N + j] - C_fromGPU[i*N + j])/C[i*N + j]) > abs(tolerance)){
		
                printf("Element C[%d][%d] (%f+i%f) and C_fromGPU[%d][%d] (%f+i%f) do not match!\n", i, j, real(C[i*N + j]), imag(C[i*N + j]), i, j, real(C_fromGPU[i*N + j]), imag(C_fromGPU[i*N + j]));
                return EXIT_FAILURE;
            }
    //        printf("Element C[%d][%d] (%f+i%f) and C_fromGPU[%d][%d] (%f+i%f)\n", i, j, real(C[i*N + j]), imag(C[i*N + j]), i, j, real(C_fromGPU[i*N + j]), imag(C_fromGPU[i*N + j]));
        }
    }

    /* Clean up and output --------------------------------------------------------------*/

    hipblasDestroy(handle);

    // Free GPU memory
    cudaErrorCheck( hipFree(d_A) );
    cudaErrorCheck( hipFree(d_B) );
    cudaErrorCheck( hipFree(d_C) );

    // Free CPU memory
    free(A);
    free(B);
    free(C);
    free(C_fromGPU);

    printf("__SUCCESS__\n");

    return 0;
}
