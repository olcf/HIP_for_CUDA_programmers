#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------------------------------
This program will fill 2 NxN matrices with random numbers, compute a matrix multiply on the CPU 
and then on the GPU, compare the values for correctness, and print _SUCCESS_ (if successful).

Written by Tom Papatheodore
------------------------------------------------------------------------------------------------*/

#include <stdio.h>
#include <essl.h>
#include <hipblas.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
        printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
        exit(0);                                                                            \
    }                                                                                     \
}while(0)

#define N 512

int main(int argc, char *argv[])
{

    // Set device to GPU 0
    cudaErrorCheck( hipSetDevice(0) );

    /* Allocate memory for A, B, C on CPU ----------------------------------------------*/
    double *A = (double*)malloc(N*N*sizeof(double));
    double *B = (double*)malloc(N*N*sizeof(double));
    double *C = (double*)malloc(N*N*sizeof(double));

    /* Set Values for A, B, C on CPU ---------------------------------------------------*/

    // Max size of random double
    double max_value = 10.0;

    // Set A, B, C
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            A[i*N + j] = (double)rand()/(double)(RAND_MAX/max_value);
            B[i*N + j] = (double)rand()/(double)(RAND_MAX/max_value);
            C[i*N + j] = 0.0;
        }
    }

    /* Allocate memory for d_A, d_B, d_C on GPU ----------------------------------------*/
    double *d_A, *d_B, *d_C;
    cudaErrorCheck( hipMalloc(&d_A, N*N*sizeof(double)) );
    cudaErrorCheck( hipMalloc(&d_B, N*N*sizeof(double)) );
    cudaErrorCheck( hipMalloc(&d_C, N*N*sizeof(double)) );

    /* Copy host arrays (A,B,C) to device arrays (d_A,d_B,d_C) -------------------------*/
    cudaErrorCheck( hipMemcpy(d_A, A, N*N*sizeof(double), hipMemcpyHostToDevice) );
    cudaErrorCheck( hipMemcpy(d_B, B, N*N*sizeof(double), hipMemcpyHostToDevice) );
    cudaErrorCheck( hipMemcpy(d_C, C, N*N*sizeof(double), hipMemcpyHostToDevice) );	

    /* Perform Matrix Multiply on CPU --------------------------------------------------*/

    const double alpha = 1.0;
    const double beta = 0.0;

    dgemm("n", "n", N, N, N, alpha, A, N, B, N, beta, C, N);

    /* Perform Matrix Multiply on GPU --------------------------------------------------*/

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasStatus_t status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
    if (status != HIPBLAS_STATUS_SUCCESS){
        printf("hipblasDgemm failed with code %d\n", status);
        return EXIT_FAILURE;
    }

	/* Copy values of d_C back from GPU and compare with values calculated on CPU ------*/

    // Copy values of d_C (computed on GPU) into host array C_fromGPU	
    double *C_fromGPU = (double*)malloc(N*N*sizeof(double));	
    cudaErrorCheck( hipMemcpy(C_fromGPU, d_C, N*N*sizeof(double), hipMemcpyDeviceToHost) );

    // Check if CPU and GPU give same results
    double tolerance = 1.0e-13;
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            if(fabs((C[i*N + j] - C_fromGPU[i*N + j])/C[i*N + j]) > tolerance){
                printf("Element C[%d][%d] (%f) and C_fromGPU[%d][%d] (%f) do not match!\n", i, j, C[i*N + j], i, j, C_fromGPU[i*N + j]);
                return EXIT_FAILURE;
            }
        }
    }

    /* Clean up and output --------------------------------------------------------------*/

    hipblasDestroy(handle);

    // Free GPU memory
    cudaErrorCheck( hipFree(d_A) );
    cudaErrorCheck( hipFree(d_B) );
    cudaErrorCheck( hipFree(d_C) );

    // Free CPU memory
    free(A);
    free(B);
    free(C);
    free(C_fromGPU);

    printf("__SUCCESS__\n");

    return 0;
}
