#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------------------------------
redundant_MM

For each MPI rank, this program does the following:
  * Fill 2 NxN matrices with random numbers
  * Compute a matrix multiply on the CPU
	* Compute a matrix multiply on the GPU (loop_count times)
  * Compare the CPU and GPU results for consistency
  * Output the total runtime and time spent computing on the GPUs for each rank (and max)
    as well as the hardware thread and GPU used on a specific node

USAGE:

Two command line arguments must be supplied:
	N (matrix size)
	loop_count (number of times hipblasDgemm is called)

For example,

	$ jsrun -n6 -c1 -g1 -a1 -r3 ./redundant_MM 2048 1000 | sort
	(N = 2048) Max Total Time: 6.879220 Max GPU Time: 2.816899
	Rank 000, HWThread 002, GPU 0, Node h41n09 - Total Time: 6.855115 GPU Time: 2.804994
	Rank 001, HWThread 004, GPU 1, Node h41n09 - Total Time: 6.816647 GPU Time: 2.814934
	Rank 002, HWThread 008, GPU 2, Node h41n09 - Total Time: 6.879220 GPU Time: 2.816899
	Rank 003, HWThread 000, GPU 0, Node h41n10 - Total Time: 5.862273 GPU Time: 2.814339
	Rank 004, HWThread 005, GPU 1, Node h41n10 - Total Time: 5.798143 GPU Time: 2.765094
	Rank 005, HWThread 010, GPU 2, Node h41n10 - Total Time: 5.746687 GPU Time: 2.785626

Written by Tom Papatheodore
------------------------------------------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sched.h>
#include <mpi.h>
#include <essl.h>
#include <hipblas.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

int main(int argc, char *argv[])
{

	/* -------------------------------------------------------------------------------------------
		MPI Initialization 
	--------------------------------------------------------------------------------------------*/
	MPI_Init(&argc, &argv);

	int size;
	MPI_Comm_size(MPI_COMM_WORLD, &size);

	int rank;
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);

	char name[MPI_MAX_PROCESSOR_NAME];
	int resultlength;
	MPI_Get_processor_name(name, &resultlength);
	
	const char* nl_rank = getenv("OMPI_COMM_WORLD_LOCAL_RANK");
	int node_local_rank = atoi(nl_rank);

	/* -------------------------------------------------------------------------------------------
		Other Initialization 
	--------------------------------------------------------------------------------------------*/

	// Start Total Runtime Timer
	double start_time, end_time, elapsed_time;
	start_time = MPI_Wtime();

	// Matrix size
	int N;

	// Number of times hipblasDgemm is called
	int loop_count;

	// Check for proper command line arguments
	if(argc != 3){
		printf("Must supply two arguments: N (matrix size) and loop_count (number of hipblasDgemm calls). Exiting...\n");
		exit(0);
	}
	else{
		for(int i=0; i<strlen(argv[1]); i++){
			if(!isdigit(argv[1][i])){
				printf("1st argument must be a positive integer! Exiting...\n");
				exit(0);
			}
		}
		N = atoi(argv[1]);		

		for(int i=0; i<strlen(argv[2]); i++){
			if(!isdigit(argv[2][i])){
				printf("2nd argument must be a positive integer! Exiting...\n");
				exit(0);
			}
		}
		loop_count = atoi(argv[2]);
	}

	// Find hardware thread being used by each MPI rank
	int hwthread = sched_getcpu();

	// Find how many GPUs CUDA runtime says are available
	int num_devices = 0;
	cudaErrorCheck( hipGetDeviceCount(&num_devices) );

	// Map MPI ranks to GPUs according to node-local MPI rank (round-robin)
	int gpu_id = node_local_rank % num_devices;
	cudaErrorCheck( hipSetDevice(gpu_id) );

	/* -------------------------------------------------------------------------------------------
		Allocate memory for arrays on CPU and GPU
	--------------------------------------------------------------------------------------------*/

	// Allocate memory for A, B, C on CPU
	double *A = (double*)malloc(N*N*sizeof(double));
	double *B = (double*)malloc(N*N*sizeof(double));
	double *C = (double*)malloc(N*N*sizeof(double));

    // Allocate memory for d_A, d_B, d_C on GPU
    double *d_A, *d_B, *d_C;
    cudaErrorCheck( hipMalloc(&d_A, N*N*sizeof(double)) );
    cudaErrorCheck( hipMalloc(&d_B, N*N*sizeof(double)) );
    cudaErrorCheck( hipMalloc(&d_C, N*N*sizeof(double)) );

    /* -------------------------------------------------------------------------------------------
        Fill arrays on CPU
    --------------------------------------------------------------------------------------------*/

	// Max size of random double
	double max_value = 10.0;

	// Set A, B, and C
	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){
			A[i*N + j] = (double)rand()/(double)(RAND_MAX/max_value);
			B[i*N + j] = (double)rand()/(double)(RAND_MAX/max_value);
			C[i*N + j] = 0.0;
		}
	}

    /* -------------------------------------------------------------------------------------------
        Transfer data from CPU to GPU
    --------------------------------------------------------------------------------------------*/

	// Copy host arrays (A,B,C) to device arrays (d_A,d_B,d_C)
	cudaErrorCheck( hipMemcpy(d_A, A, N*N*sizeof(double), hipMemcpyHostToDevice) );
	cudaErrorCheck( hipMemcpy(d_B, B, N*N*sizeof(double), hipMemcpyHostToDevice) );
	cudaErrorCheck( hipMemcpy(d_C, C, N*N*sizeof(double), hipMemcpyHostToDevice) );	

	/* -------------------------------------------------------------------------------------------
		Perform DGEMM on CPU
	--------------------------------------------------------------------------------------------*/

	const double alpha = 1.0;
	const double beta = 0.0;

	// Perform Matrix Multiply on CPU
	dgemm("n", "n", N, N, N, alpha, A, N, B, N, beta, C, N);

    /* -------------------------------------------------------------------------------------------
        Perform DGEMM on GPU (loop_count times) and time GPU execution
    --------------------------------------------------------------------------------------------*/

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipEvent_t start_gpu, end_gpu;
	hipEventCreate(&start_gpu);
	hipEventCreate(&end_gpu);

	// Start GPU timer
	hipEventRecord(start_gpu);

	for(int i=0; i<loop_count; i++){
		// Perform Matrix Multiply on GPU
		hipblasStatus_t status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
		if (status != HIPBLAS_STATUS_SUCCESS){
			printf("hipblasDgemm failed with code %d\n", status);
			return EXIT_FAILURE;
		}
	}

	// Stop GPU timer
	hipEventRecord(end_gpu);
	hipEventSynchronize(end_gpu);
	float milliseconds = 0.0;
	float seconds;

	hipEventElapsedTime(&milliseconds, start_gpu, end_gpu);
	seconds = milliseconds / 1000;

	hipblasDestroy(handle);

    /* -------------------------------------------------------------------------------------------
        Transfer results from GPU DGEMM to CPU
    --------------------------------------------------------------------------------------------*/

	// Copy values of d_C computed on GPU into host array C_fromGPU	
	double *C_fromGPU = (double*)malloc(N*N*sizeof(double));	
	cudaErrorCheck( hipMemcpy(C_fromGPU, d_C, N*N*sizeof(double), hipMemcpyDeviceToHost) );

    /* -------------------------------------------------------------------------------------------
        Check for consistency between the CPU and GPU results
    --------------------------------------------------------------------------------------------*/

	// Check if CPU and GPU give same results
	double tolerance = 1.0e-13;
	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){
			if(fabs((C[i*N + j] - C_fromGPU[i*N + j])/C[i*N + j]) > tolerance){
				printf("Element C[%d][%d] (%f) and C_fromGPU[%d][%d] (%f) do not match!\n", i, j, C[i*N + j], i, j, C_fromGPU[i*N + j]);
				return EXIT_FAILURE;
			}
		}
	}

	/* -------------------------------------------------------------------------------------------
		Clean up memory and stop timer
	--------------------------------------------------------------------------------------------*/

	// Free GPU memory
	cudaErrorCheck( hipFree(d_A) );
	cudaErrorCheck( hipFree(d_B) );
	cudaErrorCheck( hipFree(d_C) );


	// Free CPU memory
	free(A);
	free(B);
	free(C);
	free(C_fromGPU);

	// End Total Runtime Timer
	end_time = MPI_Wtime();
	elapsed_time = end_time - start_time;

	/* -------------------------------------------------------------------------------------------
		MPI Reductions to find the maximum total runtime and maximum time spent computing on GPUs.
		(These are used as proxies for total runtime and total time spent computing on GPUs)
	--------------------------------------------------------------------------------------------*/

	double total_time_max;
	MPI_Reduce(&elapsed_time, &total_time_max, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);

	float gpu_time_max;
	MPI_Reduce(&seconds, &gpu_time_max, 1, MPI_FLOAT, MPI_MAX, 0, MPI_COMM_WORLD);

	/* -------------------------------------------------------------------------------------------
		Output and finalize
	--------------------------------------------------------------------------------------------*/

	// MPI rank 0 will output the maximum total runtime and maximum time spent computing on GPUs
	if(rank == 0){
		printf("(N = %d) Max Total Time: %f Max GPU Time: %f\n", N, total_time_max, gpu_time_max);
	}

	// Each MPI rank will output its total runtime and time spent computing on GPUs
	printf("Rank %03d, HWThread %03d, GPU %d, Node %s - Total Time: %f GPU Time: %f\n", rank, hwthread, gpu_id, name, elapsed_time, seconds); 

	MPI_Finalize();

	return 0;
}
